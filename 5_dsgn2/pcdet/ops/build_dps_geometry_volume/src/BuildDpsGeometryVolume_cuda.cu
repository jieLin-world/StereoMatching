#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__device__ T bilinear_interpolate(const T* bottom_data,
    const int height, const int width,
    T y, T x) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T> 
__global__ void BuildDpsGeometryVolumeForward(const int nthreads, 
    const T* img, const T* coord, const int* disp_channels,
    const int num_batch, const int channels, const int height,
    const int width, const int sep, const int interval, const int z_num, const int y_num, const int x_num,
    T* volume) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % x_num;
    int ph = (index / x_num) % y_num;
    int pd = (index / x_num / y_num) % z_num;
    int c = (index / x_num / y_num/ z_num) % sep;
    int n = index / x_num / y_num / z_num / sep;

    T scale = (T)((x_num - 1) / interval * interval) / (x_num - 1.);

    // shift channels by the ratio of pd/maxdisp
    int c_shift = int( (T) (pw / interval * interval / scale) / (x_num - 1.) * (channels - sep + 1. - 1e-9) ); // 0 -> 32

    // AT_ASSERTM(c_shift <= (channels - sep), "c_shift is (channels - sep) at max");
    c_shift = disp_channels[c_shift];

    // compute the separated channel
    int sep_c = (c_shift / sep + 1) * sep;

    int cc;
    if ( c < c_shift + sep - sep_c )
      cc = sep_c + c;
    else 
      cc = sep_c - (sep - c);

    int index_coord_x = (((n * z_num + pd) * y_num + ph) * x_num + pw) * 2;
    int index_coord_y = index_coord_x + 1;
    T coord_y = (coord[index_coord_y] + 1.) / 2. * (height - 1.);
    T coord_x = (coord[index_coord_x] + 1.) / 2. * (width - 1.);

    const T* offset_input = img + (n * channels + cc) * height * width;
    volume[index] = bilinear_interpolate(offset_input, height, width, coord_y, coord_x);
  }
}


template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width,
    T y, T x,
    T & w1, T & w2, T & w3, T & w4,
    int & x_low, int & x_high, int & y_low, int & y_high) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int) y;
  x_low = (int) x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename T>
__global__ void BuildDpsGeometryVolumeBackwardFeature(const int nthreads, 
    const T* grad, const T* coord, const int* disp_channels,
    const int num_batch, const int channels, const int height,
    const int width, const int sep, const int interval, const int z_num, const int y_num, const int x_num,
    T* grad_input) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % x_num;
    int ph = (index / x_num) % y_num;
    int pd = (index / x_num / y_num) % z_num;
    int c = (index / x_num / y_num/ z_num) % sep;
    int n = index / x_num / y_num / z_num / sep;

    T scale = (T)((x_num - 1) / interval * interval) / (x_num - 1.);

    // shift channels by the ratio of pd/maxdisp
    int c_shift = int( (T) (pw / interval * interval / scale) / (x_num - 1.) * (channels - sep + 1. - 1e-9) ); // 0 -> 32

    // AT_ASSERTM(c_shift <= (channels - sep), "c_shift is (channels - sep) at max");
    c_shift = disp_channels[c_shift];

    // compute the separated channel
    int sep_c = (c_shift / sep + 1) * sep;

    int cc;
    if ( c < c_shift + sep - sep_c )
      cc = sep_c + c;
    else 
      cc = sep_c - (sep - c);
    
    int index_coord_x = (((n * z_num + pd) * y_num + ph) * x_num + pw) * 2;
    int index_coord_y = index_coord_x + 1;
    T coord_y = (coord[index_coord_y] + 1.) / 2. * (height - 1.);
    T coord_x = (coord[index_coord_x] + 1.) / 2. * (width - 1.);

    T w1, w2, w3, w4;
    int x_low, x_high, y_low, y_high;

    bilinear_interpolate_gradient(height, width, coord_y, coord_x,
        w1, w2, w3, w4,
        x_low, x_high, y_low, y_high);

    T top_diff_this_bin = grad[index];
    T g1 = top_diff_this_bin * w1;
    T g2 = top_diff_this_bin * w2;
    T g3 = top_diff_this_bin * w3;
    T g4 = top_diff_this_bin * w4;

    T* offset_grad_input = grad_input + (n * channels + cc) * height * width;
    if (w1 >= 1e-10)
        atomicAdd(offset_grad_input + y_low * width + x_low, static_cast<T>(g1));
    if (w2 >= 1e-10)
        atomicAdd(offset_grad_input + y_low * width + x_high, static_cast<T>(g2));
    if (w3 >= 1e-10)
        atomicAdd(offset_grad_input + y_high * width + x_low, static_cast<T>(g3));
    if (w4 >= 1e-10)
        atomicAdd(offset_grad_input + y_high * width + x_high, static_cast<T>(g4));
  } // CUDA_1D_KERNEL_LOOP
} // BuildDpsGeometryVolumeBackward


at::Tensor BuildDpsGeometryVolume_forward_cuda(const at::Tensor& img,
                                 const at::Tensor& coord,
                                 const at::Tensor& disp_channels,
                                 const int sep,
                                 const int interval) {
  AT_ASSERTM(img.type().is_cuda(), "img must be a CUDA tensor");
  AT_ASSERTM(coord.type().is_cuda(), "coord must be a CUDA tensor");

  AT_ASSERTM((img.size(0) == coord.size(0)) && (coord.size(4) == 2), \
    "Image and coord should of same batch.");

  auto num_batch = img.size(0);
  auto channels = img.size(1);
  auto height = img.size(2);
  auto width = img.size(3);
  auto z_num = coord.size(1);
  auto y_num = coord.size(2);
  auto x_num = coord.size(3);
  AT_ASSERTM(interval <= x_num - 1, "interval should be less or equal to z_num - 1");

  auto output = at::empty({num_batch, sep, z_num, y_num, x_num}, img.options());
  auto output_size = num_batch * sep * z_num * y_num * x_num;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)(output_size), 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(img.type(), "BuildDpsGeometryVolume_forward", [&] {
    BuildDpsGeometryVolumeForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         img.contiguous().data<scalar_t>(),
         coord.contiguous().data<scalar_t>(),
         disp_channels.contiguous().data<int>(),
         num_batch,
         channels,
         height,
         width,
         sep,
         interval,
         z_num,
         y_num,
         x_num,
         output.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return output;
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor BuildDpsGeometryVolume_backward_cuda(const at::Tensor& grad,
                                  const at::Tensor& coord,
                                  const at::Tensor& disp_channels,
                                  const int height,
                                  const int width,
                                  const int channels,
                                  const int sep,
                                  const int interval) {
  AT_ASSERTM(coord.type().is_cuda(), "coord must be a CUDA tensor");

  auto num_batch = grad.size(0);
  auto z_num = grad.size(2);
  auto y_num = grad.size(3);
  auto x_num = grad.size(4);
  
  auto grad_input = at::zeros({num_batch, channels, height, width}, grad.options());

  AT_ASSERTM((z_num == coord.size(1)) && (y_num == coord.size(2)) && (x_num == coord.size(3)),
      "grad shape is wrong");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "BuildDpsGeometryVolume_backward", [&] {
    BuildDpsGeometryVolumeBackwardFeature<scalar_t><<<grid, block, 0, stream>>>(
         grad.numel(),
         grad.contiguous().data<scalar_t>(),
         coord.contiguous().data<scalar_t>(),
         disp_channels.contiguous().data<int>(),
         num_batch,
         channels,
         height,
         width,
         sep,
         interval,
         z_num,
         y_num,
         x_num,
         grad_input.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}

